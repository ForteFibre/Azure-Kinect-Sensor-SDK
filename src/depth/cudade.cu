
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <float.h>
#include <stdlib.h>
#include <time.h>

#define E_PI 3.1415926535897932384626433832795028841971693993751058209749445923078164062


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

#define PROFILE 0

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__device__ static void GetPhase(const float* d, float* phase, float* amplitude, float* offset)
{
    // See https://math.stackexchange.com/questions/118526/fitting-a-sine-wave-of-known-frequency-through-three-points
    float c = (d[0] + d[2]) / 2.0f;
    *offset = c;
    //float a = sqrtf(powf(d[0] - c, 2.0f) + powf(d[1] - c, 2.0f));
    //*amplitude = a;
    float b = atan2f(d[0] - c, d[1] - c);
    *phase = b;
}


__device__ float GetNFOVData(int x, int y, int frame, const unsigned char* image)
{
    const int frame_width = 640;
    const int frame_height = 576;
    const int frame_stride = frame_width * 8 / 5;
    int offset = ((frame + 1) % 4) * frame_stride / 4;
    int block_of_8 = x / 5;
    int line_idx = offset + block_of_8 * 8 + x % 5;
    int idx = y * frame_stride + frame * frame_height * frame_stride + line_idx;

    int d = (int)image[idx];
    if (d >= 64)
        d = 64 - d;
    return d;
}

__device__ static inline float GetNFOVDistance(const float* phases, float* err)
{
    /* Calibration gives us:
        d1 = 0.734 * phase1 - 0.300 
        d2 = 0.778 * phase2 - 0.150
        d3 = 2.866 * phase3 - 1.053
        
        For max dist of 3.86m (as per data sheet), we get
        max phase1 = 5.66 * 2pi
        max phase2 = 5.42 * 2pi
        max phase3 = 1.71 * 2pi */

    const int f1n = 5;
    const int f2n = 5;
    const int f3n = 1;

    float best_err = FLT_MAX;
    float best_dist = 0.0f;

    int best_i = 0;
    int best_j = 0;
    int best_k = 0;

    // brute force algorithm as per https://medium.com/chronoptics-time-of-flight/phase-wrapping-and-its-solution-in-time-of-flight-depth-sensing-493aa8b21c42
    for (int k = 0; k <= f3n; k++)
    {
        for (int j = 0; j <= f2n; j++)
        {
            for (int i = 0; i <= f1n; i++)
            {
                //float d1 = (phases[0] + (float)i * 2.0f * E_PI) / 2.0f / f1;
                //float d2 = (phases[1] + (float)j * 2.0f * E_PI) / 2.0f / f2;
                //float d3 = (phases[2] + (float)k * 2.0f * E_PI) / 2.0f / f3;

                float d1 = 0.734f / 2.0f / E_PI * (phases[0] + (float)i * 2.0f * E_PI) - 0.300f;
                float d2 = 0.778f / 2.0f / E_PI * (phases[1] + (float)j * 2.0f * E_PI) - 0.357f;
                float d3 = 2.866f / 2.0f / E_PI * (phases[2] + (float)k * 2.0f * E_PI) - 1.053f;

                float d_mean = (d1 + d2 + d3) / 3.0f;
                //float d_var = (powf(d1 - d_mean, 2.0f) + powf(d2 - d_mean, 2.0f) + powf(d3 - d_mean, 2.0f)) / 3.0f;
                //float d_var = fabsf(d1 - d_mean) + fabsf(d2 - d_mean) + fabsf(d3 - d_mean);
                float d_var = ((d1 - d_mean) * (d1 - d_mean) + (d2 - d_mean) * (d2 - d_mean) + (d3 - d_mean) * (d3 - d_mean)) / 3.0f;
                //printf("%i,%i,%i: %f,%f,%f (%f)\n",
                //    i, j, k, d1, d2, d3, sq_err);
                // TODO: profile to see which of these is best
#if 0
                if (d_var < best_err)
                {
                    best_err = d_var;
                    best_dist = d_mean;
                    best_i = i;
                    best_j = j;
                    best_k = k;
                }
#endif

#if 1
                best_dist = d_var < best_err ? d_mean : best_dist;
                best_i = d_var < best_err ? i : best_i;
                best_j = d_var < best_err ? j : best_j;
                best_k = d_var < best_err ? k : best_k;
                best_err = d_var < best_err ? d_var : best_err;
#endif
            }
        }
    }

    if (err)
    {
        *err = best_err;
    }

    //best_dist *= 300.0f / 2.0f / E_PI;      // c / 10e6 to account for freq in MHz

    (void)best_i;
    (void)best_j;
    (void)best_k;
    //printf("%i,%i,%i: %f (%f)\n", best_i, best_j, best_k, best_dist, best_err);

    return best_dist;
}

#if PROFILE
#define PROFILE_START(a) unsigned int pstart ## a, pend ## a; pstart ## a = clock();
#define PROFILE_END(a) pend ## a = clock(); dev_times ## a ## [outidx] = pend ## a - pstart ## a;
#else
#define PROFILE_START(a)
#define PROFILE_END(a)
#endif


__global__ void NFOVUnbinnedKernel(unsigned short int* depth_out,
    unsigned short int* ir_out,
    const unsigned char* data
#if PROFILE    
    , unsigned int *dev_times1, unsigned int *dev_times2, unsigned int *dev_times3
#endif
    )
{
    int outidx = threadIdx.x + blockIdx.x * blockDim.x;

    const int frame_width = 640;

    int x = outidx % frame_width;
    int y = outidx / frame_width;

    float phases[3];
    float offsets[3];
    float amplitudes[3];
    float d[9];

    PROFILE_START(1);
    PROFILE_START(2);
    for (int i = 0; i < 9; i++)
    {
        d[i] = GetNFOVData(x, y, i, data);
    }

    for (int i = 0; i < 3; i++)
    {
        GetPhase(&d[i * 3], &phases[i], &amplitudes[i], &offsets[i]);
    }
    PROFILE_END(2);

    // Apply a fiddle factor based upon experimentation to account for time delay
    //  between imaging each column of the IR image
    phases[0] = fmodf(phases[0] - 2.7f * (float)x / 200.0f, E_PI * 2.0f);
    phases[1] = fmodf(phases[1] - 2.55f * (float)x / 200.0f, E_PI * 2.0f);
    phases[2] = fmodf(phases[2] - 1.05f * (float)x / 200.0f, E_PI * 2.0f);
    if (phases[0] < 0.0f) phases[0] += E_PI * 2.0f;
    if (phases[1] < 0.0f) phases[1] += E_PI * 2.0f;
    if (phases[2] < 0.0f) phases[2] += E_PI * 2.0f;

    PROFILE_START(3);
    float dist = GetNFOVDistance(phases, NULL);
    PROFILE_END(3);
    float irf = fabsf((offsets[0] + offsets[1] + offsets[2]) / 3.0f / dist / dist * 1000.0f);

    unsigned short int depth_val = (unsigned short int)(dist * 1000.0f); // mm distance
    unsigned short int ir_val = (unsigned short int)irf;

    depth_out[outidx] = depth_val;
    ir_out[outidx] = ir_val;

    PROFILE_END(1);
}

// buffers to hold device data
unsigned char* dev_data;
unsigned short* dev_ir_out;
unsigned short* dev_depth_out;

unsigned int* dev_times1;
unsigned int* dev_times2;
unsigned int* dev_times3;

// buffer sizes
const int NFOVUnbinned_in_count = 1024 * 576 * 9;
const int NFOVUnbinned_out_count = 640 * 576;

const int nthreads = 128;

extern "C" {

// Function to call the kernel
void RunNFOVUnbinnedCalculation(unsigned short int* depth_out,
    unsigned short int* ir_out,
    const unsigned char* data)
{
    // TODO: add error checking here
    hipError_t cudaStatus = hipMemcpy(dev_data, data, NFOVUnbinned_in_count * sizeof(unsigned char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return;
    }

    NFOVUnbinnedKernel <<<NFOVUnbinned_out_count / nthreads, nthreads>>> (dev_depth_out, dev_ir_out, dev_data
#if PROFILE
        , dev_times1, dev_times2, dev_times3
#endif
        );

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return;
    }

    cudaStatus = hipMemcpy(depth_out, dev_depth_out, NFOVUnbinned_out_count * sizeof(unsigned short), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return;
    }
    cudaStatus = hipMemcpy(ir_out, dev_ir_out, NFOVUnbinned_out_count * sizeof(unsigned short), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return;
    }

#if PROFILE
    unsigned int* times1 = (unsigned int*)malloc(NFOVUnbinned_out_count * sizeof(unsigned int));
    unsigned int* times2 = (unsigned int*)malloc(NFOVUnbinned_out_count * sizeof(unsigned int));
    unsigned int* times3 = (unsigned int*)malloc(NFOVUnbinned_out_count * sizeof(unsigned int));
    cudaMemcpy(times1, dev_times1, NFOVUnbinned_out_count * sizeof(unsigned int), cudaMemcpyDeviceToHost);
    cudaMemcpy(times2, dev_times2, NFOVUnbinned_out_count * sizeof(unsigned int), cudaMemcpyDeviceToHost);
    cudaMemcpy(times3, dev_times3, NFOVUnbinned_out_count * sizeof(unsigned int), cudaMemcpyDeviceToHost);

    unsigned int times1_worst = 0;
    unsigned int times2_worst = 0;
    unsigned int times3_worst = 0;

    for (int i = 0; i < NFOVUnbinned_out_count; i++)
    {
        if (times1[i] >= times1_worst) times1_worst = times1[i];
        if (times2[i] >= times2_worst) times2_worst = times2[i];
        if (times3[i] >= times3_worst) times3_worst = times3[i];
    }

    printf("t1: %i, t2: %i, t3: %i\n", times1_worst, times2_worst, times3_worst);

    free(times1);
    free(times2);
    free(times3);
#endif
}

// Init function
void InitNFOVUnbinnedCalculation()
{
    hipSetDevice(0);
    hipMalloc(&dev_data, NFOVUnbinned_in_count * sizeof(unsigned char));
    hipMalloc(&dev_ir_out, NFOVUnbinned_out_count * sizeof(unsigned short int));
    hipMalloc(&dev_depth_out, NFOVUnbinned_out_count * sizeof(unsigned short int));

#ifdef PROFILE
    hipMalloc(&dev_times1, NFOVUnbinned_out_count * sizeof(unsigned int));
    hipMalloc(&dev_times2, NFOVUnbinned_out_count * sizeof(unsigned int));
    hipMalloc(&dev_times3, NFOVUnbinned_out_count * sizeof(unsigned int));
#endif
}

// Dealloc function
void DeinitNFOVUnbinnedCalculation()
{
    if (dev_data)
    {
        hipFree(dev_data);
        dev_data = NULL;
    }
    if (dev_ir_out)
    {
        hipFree(dev_ir_out);
        dev_ir_out = NULL;
    }
    if (dev_depth_out)
    {
        hipFree(dev_depth_out);
        dev_depth_out = NULL;
    }

#if PROFILE
    if (dev_times1)
    {
        cudaFree(dev_times1);
        dev_times1 = NULL;
    }
    if (dev_times2)
    {
        cudaFree(dev_times2);
        dev_times2 = NULL;
    }
    if (dev_times3)
    {
        cudaFree(dev_times3);
        dev_times3 = NULL;
    }
#endif

}

}
